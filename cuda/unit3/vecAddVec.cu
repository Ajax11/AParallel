#include "hip/hip_runtime.h"


__global__ 
void vecAddKernel(float* A, float* B, float* C, int n){
	/* the  input is A the output matrix, B vector, c vector, n size */
	int i = threadIdx.x + blockDim.x * blockIdx.x;
	if(i < n*n)
		A[i] = B[i] + C[i];
}


__host__
void vecAdd(float* A, float* B, float* C, int n)
{
	int size = n * sizeof(float);
	float *d_A, *d_B, *d_C;
	hipMalloc((void **) &d_A, size);
	hipMemcpy(d_A, A, size, hipMemcpyHostToDevice);
	hipMalloc((void **) &B_d, size);
	hipMemcpy(d_B, B, size, hipMemcpyHostToDevice);
	hipMalloc((void **) &d_C, size);
	vecAddKernel<<<ceil(n/256.0), 256>>>(d_A, d_B, d_C, n);
	hipMemcpy(C, d_C, size, hipMemcpyDeviceToHost);
	// Free device memory for A, B, C
	hipFree(d_A);
	hipFree(d_B);
	hipFree(d_C);
}

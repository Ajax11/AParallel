
#include <iostream>
#include <stdio.h>
#include <hip/hip_runtime.h>
#define SIZEM 1000

__global__ 
void matrixXvecKernel(float* A, float* B, float* C, int n){
	/* the  input is A the output matrix, B matrix, c vector, n size */
	//int size = n * sizeof(float);

	int j = threadIdx.x + blockDim.x * blockIdx.x;
	if( j < n ){
		A[j] = 0;
		for (int i = 0; i < n; ++i){
			A[j] += B[ ( j * n ) + i ] * C[i];
		}
	}
}

__host__
void product(float* A, float* B, float* C, int n){
	int size = n * sizeof(float);
	
	float* d_A;
	float* d_B;
	float* d_C;

	hipMalloc((void **) &d_B, size * n);
	hipMemcpy(d_B, B, size*n, hipMemcpyHostToDevice);

	hipMalloc((void **) &d_C, size);
	hipMemcpy(d_C, C, size, hipMemcpyHostToDevice);

	hipMalloc((void **) &d_A, size);
	hipMemcpy(d_A, A, size, hipMemcpyHostToDevice);

	matrixXvecKernel<<<ceil(n/256.0), 256>>>(d_A, d_B, d_C, n);
	hipMemcpy(A, d_A, size, hipMemcpyDeviceToHost);

	// Free device memory for A, B, C
	hipFree(d_A);
	hipFree(d_B);
	hipFree(d_C);
}


__host__
int main(int argc, char const *argv[]){

	float* a;
	float* b;
	float* c;

	a = (float*) malloc(SIZEM*sizeof(float));
	b = (float*) malloc(SIZEM*SIZEM*sizeof(float));
	c = (float*) malloc(SIZEM*sizeof(float));

	for (int i = 0; i < SIZEM; ++i){
		a[i] = 0;
	}

	for (int i = 0; i < SIZEM*SIZEM; ++i){
		b[i] = 1;
	}

	for (int i = 0; i < SIZEM; ++i){
		c[i] = 1;
	}

	product(a, b, c, SIZEM);
	for (int i = 0; i < SIZEM; ++i){
		if (i % SIZEM == 0){
			std::cout << '\n';
		}
		std::cout << a[i] << '\t';
	}
			std::cout << '\n';

	return 0;
}


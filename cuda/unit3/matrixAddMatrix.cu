#include <iostream>
#include <stdio.h>
#include <hip/hip_runtime.h>
#define SIZEM 100

__global__ 
void matrixAddKernel(float* A, float* B, float* C, int n){
	/* the  input is A the output matrix, B matrix, c matrix, n size */
	int i = threadIdx.x + blockDim.x * blockIdx.x;
	if(i < (n*n)){
		A[i] = B[i] + C[i];
	}
}

__global__ 
void matrixAddKernelRow(float* A, float* B, float* C, int n){
	/* the  input is A the output matrix, B matrix, c matrix, n size */
	int j = threadIdx.x + blockDim.x * blockIdx.x;
	if(j < n)
		for (int i = 0; i < n; ++i){
			A[ ( j * n ) + i ] = B[ ( j * n ) + i ] + C[ ( j * n ) + i ];
		}
}

__global__ 
void matrixAddKernelColumn(float* A, float* B, float* C, int n){
	/* the  input is A the output matrix, B matrix, c matrix, n size */
	int i = threadIdx.x + blockDim.x * blockIdx.x;
	if(i < n)
		for (int j = 0; j < n; ++j){
			A[ ( j * n ) + i ] = B[ ( j * n ) + i ] + C[ ( j * n ) + i ];
		}
}

__host__
void matrixAdd(float* A, float* B, float* C, int n){
	int size = n * n * sizeof(float);
	
	float* d_A;
	float* d_B;
	float* d_C;
	
	hipMalloc((void **) &d_B, size);
	hipMemcpy(d_B, B, size, hipMemcpyHostToDevice);

	hipMalloc((void **) &d_C, size);
	hipMemcpy(d_C, C, size, hipMemcpyHostToDevice);

	hipMalloc((void **) &d_A, size);
	hipMemcpy(d_A, A, size, hipMemcpyHostToDevice);

	matrixAddKernelColumn<<<ceil((n*n)/256.0), 256>>>(d_A, d_B, d_C, n);
	hipMemcpy(A, d_A, size, hipMemcpyDeviceToHost);


	// Free device memory for A, B, C
	hipFree(d_A);
	hipFree(d_B);
	hipFree(d_C);
}


__host__
int main(int argc, char const *argv[]){
	/*
	float a[SIZEM];
	float b[SIZEM*SIZEM];
	float c[SIZEM];
	*/
	float* a;
	float* b;
	float* c;

	a = (float*) malloc(SIZEM*SIZEM*sizeof(float));
	b = (float*) malloc(SIZEM*SIZEM*sizeof(float));
	c = (float*) malloc(SIZEM*SIZEM*sizeof(float));


	for (long int i = 0; i < SIZEM*SIZEM; ++i){
		a[i] = 0;
	}

	for (long int i = 0; i < SIZEM*SIZEM; ++i){
		b[i] = 1;
	}

	for (long int i = 0; i < SIZEM*SIZEM; ++i){
		c[i] = 1;
	}

	matrixAdd(a, b, c, SIZEM);

	for (int i = 0; i < SIZEM*SIZEM; ++i){
		if (i % SIZEM == 0){
			std::cout << '\n';
		}
		std::cout << a[i] << '\t';
	}
			std::cout << '\n';

	return 0;
}

/*

	

*/

